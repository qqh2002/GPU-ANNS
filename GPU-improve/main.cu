#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string>
#include<set>
#include<hip/hip_runtime.h>
#include<fstream>
#include<sstream>
#include<iostream>
#include"parser.h"
#include"kernel_search.h"

float* data=NULL;
int* graph=NULL;
int* graph_=NULL;
int* result=NULL;
int*  cycle_times=NULL;
std::vector<float> query_vector;
float* query=NULL;
std::vector<std::vector<int>> cmp;

int num_vertices=PLACE_HOLDER_VERTICES;
int vertex_offset_shift=5;
int dim=PLACE_HOLDER_DIM;
int len_q=0;
int num_of_candidates=PLACE_HOLDER_CANDIDATES;
int num_of_topk=PLACE_HOLDER_DISPLAY;
int num_of_explored_points=0;
int num_of_topk_=0;
int width = PLACE_HOLDER_WIDTH;

void load_graph(std::string file = "bfsg.graph")
{
    FILE* fp = fopen(file.c_str(),"rb");
    auto cnt = fread(graph,sizeof(int) * (num_vertices << vertex_offset_shift),1,fp);
    fclose(fp);
}

void load_data(std::string file = "bfsg.data")
{
    FILE* fp = fopen(file.c_str(),"rb");
    auto cnt = fread(data,sizeof(float) * (num_vertices*dim),1,fp);
    fclose(fp);
}

void query_callback(int idx,std::vector<std::pair<int,float>> point){
    for(int i=0;i<point.size();i++)
    {
        query_vector.push_back(point[i].second);
    }
    len_q++;
}

void compute_recall(std::vector<std::vector<int>> compare)
{
    double recall;
    int ans=0,sum=0;
    std::ifstream file("ans.txt");
    std::vector<std::vector<int>> data;
    std::string line;

    if (!file.is_open()) {
        std::cout << "Unable to open file" << std::endl;
        return ;
    }

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        std::vector<int> row;
        int value;

        while (iss >> value) {
            row.push_back(value);
        }

        data.push_back(row);
    }

    file.close();
    for(int i=0;i<compare.size()&&i<data.size();i++)
    {
        std::vector<int> r;
        std::vector<int> a;
        for(int j=0;j< num_of_topk;j++)
        {
            sum++;
            r.push_back(compare[i][j]);
            a.push_back(data[i][j]);
        }
        for(int j=0;j<r.size();j++)
        {
            for(int t=0;t<a.size();t++)
            {
                if(r[j]==a[t])
                {
                    ans++;
                }
            }
        }
    }
    recall=ans/double(sum);
    fprintf(stderr,"recall:%lf ans:%d sum:%d\n",recall,ans,sum);
}

int main()
{
    hipSetDevice(0);
    graph=(int*)malloc(sizeof(int)*(num_vertices << vertex_offset_shift));
    data=(float*)malloc(sizeof(float)*(num_vertices * dim));
    load_graph();
    load_data();
    graph_=(int*)malloc(sizeof(int)*(num_vertices << vertex_offset_shift));
    for(int i = 0;i < num_vertices ;i++)
    {
        int len = graph[i<<vertex_offset_shift];
        for(int j = 0;j < (1 << vertex_offset_shift);j++)
        {
            if(j == 0)
            {
                graph_[i*(1<<vertex_offset_shift)+j]=num_vertices;
                continue;
            }
            if(j<=len)
            {
                graph_[i*(1<<vertex_offset_shift)+j] = graph[i*(1<<vertex_offset_shift)+j];
                
            }else{
                graph_[i*(1<<vertex_offset_shift)+j] = num_vertices;
            }
        }
    }
    free(graph);

    std::unique_ptr<Parser> query_parser(new Parser("test.txt",query_callback));
    query=(float*)malloc(sizeof(float)*(len_q*dim));
    for(int i=0;i<(len_q*dim);i++)
    {
        query[i]=query_vector[i];
    }

    num_of_topk_ = pow(2.0, ceil(log(num_of_topk) / log(2)));
    num_of_explored_points = num_of_candidates;
    num_of_candidates = pow(2.0, ceil(log(num_of_candidates) / log(2)));

    float* d_data;
    hipMalloc(&d_data, sizeof(float) * num_vertices * dim);
    hipMemcpy(d_data, data, sizeof(float) * num_vertices * dim, hipMemcpyHostToDevice);

    float* d_query;
    hipMalloc(&d_query, sizeof(float) * len_q * dim);
    hipMemcpy(d_query, query, sizeof(float) * len_q * dim, hipMemcpyHostToDevice);

    int* d_result;
    hipMalloc(&d_result, sizeof(int) * len_q * num_of_topk_);
    result=(int*)malloc(sizeof(int) * len_q * num_of_topk_);

    int* d_graph;
    hipMalloc(&d_graph, sizeof(int) * (num_vertices << vertex_offset_shift));
    hipMemcpy(d_graph, graph_, sizeof(int) * (num_vertices << vertex_offset_shift), hipMemcpyHostToDevice);

    int* d_cycle;
    hipMalloc(&d_cycle,sizeof(int) * len_q);
    cycle_times = (int*)malloc(sizeof(int) * len_q);

    unsigned long long* h_time_breakdown;
    unsigned long long* d_time_breakdown;
    int num_of_phases = 6;
    hipHostMalloc(&h_time_breakdown, len_q * num_of_phases * sizeof(unsigned long long));
    hipMalloc(&d_time_breakdown, len_q * num_of_phases * sizeof(unsigned long long));
    hipMemset(d_time_breakdown, 0, len_q * num_of_phases * sizeof(unsigned long long));

    std::chrono::steady_clock::time_point kernel_begin = std::chrono::steady_clock::now();

    SearchDevice<<<len_q, 32, ((1 << vertex_offset_shift) * width + num_of_candidates) * (sizeof(KernelPair<float, int>) + sizeof(int))>>>(d_data, d_query, d_result, d_graph, num_vertices, 
                                                                                                                        len_q, vertex_offset_shift, num_of_candidates, num_of_topk_, 
                                                                                                                        num_of_explored_points, width, d_time_breakdown, d_cycle);
    
    hipDeviceSynchronize();

	std::chrono::steady_clock::time_point kernel_end = std::chrono::steady_clock::now();
    printf("kernel takes %ld microseconds\n",std::chrono::duration_cast<std::chrono::microseconds>(kernel_end - kernel_begin).count());                                                                                                             
    fprintf(stderr,"kernel takes %ld microseconds\n",std::chrono::duration_cast<std::chrono::microseconds>(kernel_end - kernel_begin).count());                                                                                                             

    hipMemcpy(result, d_result, sizeof(int) * len_q * num_of_topk_, hipMemcpyDeviceToHost);
    hipMemcpy(cycle_times, d_cycle, sizeof(int) * len_q, hipMemcpyDeviceToHost);
    hipMemcpy(h_time_breakdown, d_time_breakdown, len_q * num_of_phases * sizeof(unsigned long long), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    unsigned long long stage_1 = 0;
	unsigned long long stage_2 = 0;
	unsigned long long stage_3 = 0;
	unsigned long long stage_4 = 0;
	unsigned long long stage_5 = 0;
	unsigned long long stage_6 = 0;

    for (int i = 0; i < len_q; i++) {
        stage_1	+= h_time_breakdown[i * num_of_phases];
        stage_2	+= h_time_breakdown[i * num_of_phases + 1];
        stage_3	+= h_time_breakdown[i * num_of_phases + 2];
        stage_4	+= h_time_breakdown[i * num_of_phases + 3];
        stage_5	+= h_time_breakdown[i * num_of_phases + 4];
        stage_6	+= h_time_breakdown[i * num_of_phases + 5];
    }

    unsigned long long sum_of_all_stages = stage_1 + stage_2 + stage_3 + stage_4 + stage_5 + stage_6;
    fprintf(stderr,"stages percentage: %lf %lf %lf\n",
                                       (double)(stage_1+stage_2) / sum_of_all_stages,
                                       (double)(stage_3) / sum_of_all_stages,
                                       (double)(stage_4+stage_5+stage_6) / sum_of_all_stages);
    // fprintf(stderr,"stages percentage: %lf %lf %lf %lf %lf %lf\n",
    //                                    (double)(stage_1) / sum_of_all_stages,
    //                                    (double)(stage_2) / sum_of_all_stages,
    //                                    (double)(stage_3) / sum_of_all_stages,
    //                                    (double)(stage_4) / sum_of_all_stages,
    //                                    (double)(stage_5) / sum_of_all_stages,
    //                                    (double)(stage_6) / sum_of_all_stages);
    // std::cout << "stages percentage: " << (double)(stage_1) / sum_of_all_stages << " "
    //                                 << (double)(stage_2) / sum_of_all_stages << " "
    //                                 << (double)(stage_3) / sum_of_all_stages << " "
    //                                 << (double)(stage_4) / sum_of_all_stages << " "
    //                                 << (double)(stage_5) / sum_of_all_stages << " "
    //                                 << (double)(stage_6) / sum_of_all_stages << std::endl;

    int sum_cycle = 0;
    for(int i=0;i<len_q;i++)
    {
        std::vector<int> tmp;
        sum_cycle += cycle_times[i];
        for(int j=0;j<num_of_topk;j++)
        {
            tmp.push_back(result[i*num_of_topk_+j]);
        }
        cmp.push_back(tmp);
    }
    fprintf(stderr,"avg_cycle_times:%f\n",sum_cycle/(len_q*1.0));
    compute_recall(cmp);
    free(graph_);
    free(data);
    free(query);
    free(result);
    return 0;
}